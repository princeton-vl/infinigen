#include "hip/hip_runtime.h"
__global__ void upsidedown_mountains_kernel(
    size_t size,
    float3_nonbuiltin *position,
    float *sdfs,
    int *i_params,
    float *f_params
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
    }
}

extern "C" {


    void call(
        size_t size,
        float3_nonbuiltin *positions,
    ) {
        using namespace data;
        float3_nonbuiltin *d_positions;
        hipMalloc((void **)&d_positions, size * sizeof(float3_nonbuiltin));
        hipMemcpy(d_positions, positions, size * sizeof(float3_nonbuiltin), hipMemcpyHostToDevice);
        float *d_sdfs;
        hipMalloc((void **)&d_sdfs, size * sizeof(float));
        hipMemcpy(sdfs, d_sdfs, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_positions);
        hipFree(d_sdfs);
    }

}