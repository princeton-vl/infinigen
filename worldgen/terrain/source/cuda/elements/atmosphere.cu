#include "hip/hip_runtime.h"
// Copyright (c) Princeton University.
// This source code is licensed under the BSD 3-Clause license found in the LICENSE file in the root directory of this source tree.

// Authors: Zeyu Ma


#include "header.h"


__global__ void atmosphere_kernel(
    size_t size,
    float3_nonbuiltin *position,
    float *sdfs,
    int has_water,
    int *i_params, float *f_params,
    int *waterbody_i_params, float *waterbody_f_params
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        atmosphere(position[idx], sdfs + idx, has_water, i_params, f_params, waterbody_i_params, waterbody_f_params);
    }
}



extern "C" {
    void call(
        size_t size,
        float3_nonbuiltin *positions,
        float *sdfs
    ) {
        using namespace data;
        float3_nonbuiltin *d_positions;
        hipMalloc((void **)&d_positions, size * sizeof(float3_nonbuiltin));
        hipMemcpy(d_positions, positions, size * sizeof(float3_nonbuiltin), hipMemcpyHostToDevice);
        float *d_sdfs;
        hipMalloc((void **)&d_sdfs, size * sizeof(float));

        atmosphere_kernel<<<ceil(size / 256.0), 256>>>(
            size, d_positions, d_sdfs, meta_param,
            d_i_params, d_f_params, second_d_i_params, second_d_f_params
        );

        hipMemcpy(sdfs, d_sdfs, size * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_positions);
        hipFree(d_sdfs);
    }
}