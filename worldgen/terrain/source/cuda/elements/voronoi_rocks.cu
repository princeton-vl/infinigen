#include "hip/hip_runtime.h"
// Copyright (c) Princeton University.
// This source code is licensed under the BSD 3-Clause license found in the LICENSE file in the root directory of this source tree.

// Authors: Zeyu Ma


#include "header.h"


__global__ void voronoi_rocks_kernel(
    size_t size,
    float3_nonbuiltin *positions,
    float *sdfs,
    float *auxs,
    int atatchment,
    int is_caved,
    int *i_params, float *f_params,
    int *attachment_i_params, float *attachment_f_params,
    int *caves_i_params, float *caves_f_params
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int n_auxiliaries = 2;
        if (auxs == NULL) n_auxiliaries = 0;
        voronoi_rocks(
            positions[idx], sdfs + idx, auxs + idx * n_auxiliaries, atatchment, is_caved,
            i_params, f_params, attachment_i_params, attachment_f_params, caves_i_params, caves_f_params
        );
    }
}




extern "C" {

    void call(
        size_t size,
        float3_nonbuiltin *positions,
        float *sdfs,
        float *auxs
    ) {
        using namespace data;
        const int n_auxiliaries = 2;
        float3_nonbuiltin *d_positions;
        hipMalloc((void **)&d_positions, size * sizeof(float3_nonbuiltin));
        hipMemcpy(d_positions, positions, size * sizeof(float3_nonbuiltin), hipMemcpyHostToDevice);
        float *d_sdfs;
        hipMalloc((void **)&d_sdfs, size * sizeof(float));
        float *d_auxs=NULL;
        if (auxs != NULL) hipMalloc((void **)&d_auxs, size * n_auxiliaries * sizeof(float));
        voronoi_rocks_kernel<<<ceil(size / 256.0), 256>>>(
            size, d_positions, d_sdfs, d_auxs, meta_param, second_meta_param,
            d_i_params, d_f_params, second_d_i_params, second_d_f_params, third_d_i_params, third_d_f_params
        );
        hipMemcpy(sdfs, d_sdfs, size * sizeof(float), hipMemcpyDeviceToHost);
        if (auxs != NULL) hipMemcpy(auxs, d_auxs, size * n_auxiliaries * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_positions);
        if (auxs != NULL) hipFree(d_auxs);
        hipFree(d_sdfs);
    }

}