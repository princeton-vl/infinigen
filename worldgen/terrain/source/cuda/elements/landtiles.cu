#include "hip/hip_runtime.h"
// Copyright (c) Princeton University.
// This source code is licensed under the BSD 3-Clause license found in the LICENSE file in the root directory of this source tree.

// Authors: Zeyu Ma


#include "header.h"


__global__ void landtiles_kernel(
    size_t size,
    float3_nonbuiltin *position,
    float *sdfs,
    float *auxs,
    int is_caved,
    int *i_params, float *f_params,
    int *caves_i_params, float *caves_f_params
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int n_auxiliaries = 3;
        if (auxs == NULL) n_auxiliaries = 0;
        landtiles(
            position[idx], sdfs + idx, auxs + idx * n_auxiliaries, is_caved, i_params, f_params, caves_i_params, caves_f_params
        );
    }
}



extern "C" {


    void call(
        size_t size,
        float3_nonbuiltin *positions,
        float *sdfs,
        float *auxs
    ) {
        using namespace data;
        const int n_auxiliaries = 3;
        float3_nonbuiltin *d_positions;
        hipMalloc((void **)&d_positions, size * sizeof(float3_nonbuiltin));
        hipMemcpy(d_positions, positions, size * sizeof(float3_nonbuiltin), hipMemcpyHostToDevice);
        float *d_sdfs;
        hipMalloc((void **)&d_sdfs, size * sizeof(float));
        float *d_auxs=NULL;
        if (auxs != NULL) hipMalloc((void **)&d_auxs, size * n_auxiliaries * sizeof(float));
        landtiles_kernel<<<ceil(size / 256.0), 256>>>(
            size, d_positions, d_sdfs, d_auxs, meta_param,
            d_i_params, d_f_params, second_d_i_params, second_d_f_params
        );
        hipMemcpy(sdfs, d_sdfs, size * sizeof(float), hipMemcpyDeviceToHost);
        if (auxs != NULL) hipMemcpy(auxs, d_auxs, size * n_auxiliaries * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_positions);
        if (auxs != NULL) hipFree(d_auxs);
        hipFree(d_sdfs);
    }

}