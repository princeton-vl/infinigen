
#include <hip/hip_runtime.h>
// Copyright (c) Princeton University.
// This source code is licensed under the BSD 3-Clause license found in the LICENSE file in the root directory of this source tree.

// Authors: Zeyu Ma


namespace data {
    int meta_param, second_meta_param;
    int *d_i_params=NULL, *second_d_i_params=NULL, *third_d_i_params=NULL;
    float *d_f_params=NULL, *second_d_f_params=NULL, *third_d_f_params=NULL;
}

extern "C" {
    void init(
        int meta_param_, int second_meta_param_,
        size_t i_size, int *i_params, size_t f_size, float *f_params,
        size_t second_i_size, int *second_i_params, size_t second_f_size, float *second_f_params,
        size_t third_i_size, int *third_i_params, size_t third_f_size, float *third_f_params
    ) {
        using namespace data;
        meta_param = meta_param_;
        second_meta_param = second_meta_param_;
        if (i_size > 0) {
            hipMalloc((void **)&d_i_params, i_size * sizeof(int));
            hipMemcpy(d_i_params, i_params, i_size * sizeof(int), hipMemcpyHostToDevice);
        }
        if (f_size > 0) {
            hipMalloc((void **)&d_f_params, f_size * sizeof(float));
            hipMemcpy(d_f_params, f_params, f_size * sizeof(float), hipMemcpyHostToDevice);
        }
        if (second_i_size > 0) {
            hipMalloc((void **)&second_d_i_params, second_i_size * sizeof(int));
            hipMemcpy(second_d_i_params, second_i_params, second_i_size * sizeof(int), hipMemcpyHostToDevice);
        }
        if (second_f_size > 0) {
            hipMalloc((void **)&second_d_f_params, second_f_size * sizeof(float));
            hipMemcpy(second_d_f_params, second_f_params, second_f_size * sizeof(float), hipMemcpyHostToDevice);
        }
        if (third_i_size > 0) {
            hipMalloc((void **)&third_d_i_params, third_i_size * sizeof(int));
            hipMemcpy(third_d_i_params, third_i_params, third_i_size * sizeof(int), hipMemcpyHostToDevice);
        }
        if (third_f_size > 0) {
            hipMalloc((void **)&third_d_f_params, third_f_size * sizeof(float));
            hipMemcpy(third_d_f_params, third_f_params, third_f_size * sizeof(float), hipMemcpyHostToDevice);
        }
    }

    void cleanup() {
        using namespace data;
        if (d_i_params != NULL) hipFree(d_i_params);
        if (d_f_params != NULL) hipFree(d_f_params);
        if (second_d_i_params != NULL) hipFree(second_d_i_params);
        if (second_d_f_params != NULL) hipFree(second_d_f_params);
        if (third_d_i_params != NULL) hipFree(third_d_i_params);
        if (third_d_f_params != NULL) hipFree(third_d_f_params);
    }


}